#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdint.h>
//#include <helper_functions.h>
//#include <helper_cuda.h>

#define handle_error(ans) { cuda_assert((ans), __FILE__, __LINE__); }
#define handle_fft(ans) { fft_assert((ans), __FILE__, __LINE__); }
inline void cuda_assert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

inline void fft_assert(hipfftResult code, const char *file, int line, bool abort=true)
{
	if (code != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "GPUassert: FFT error %s %d\n", file, line);
		if (abort) exit(code);
	}
}

uint32_t c2c_fft(const float *input, float *output, uint32_t size)
{
	hipfftHandle plan;
	float2 *host_input = NULL;
	float2 *host_output = NULL;
	float2 *device_input = NULL;
	float2 *device_output = NULL;
	uint32_t i;
	handle_error(hipHostMalloc(&host_input, sizeof(float2) * size, hipHostMallocDefault));
	handle_error(hipHostMalloc(&host_output, sizeof(float2) * size, hipHostMallocDefault));
	for (i = 0; i < size; i++)
	{
		host_input[i].x = input[i];
		host_input[i].y = 0;
	}
	handle_error(hipMalloc(&device_input, sizeof(float2) * size));
	handle_error(hipMalloc(&device_output, sizeof(float2) * size));
	handle_error(hipMemcpy(device_input, host_input, sizeof(float2) * size, hipMemcpyHostToDevice));
	handle_fft(hipfftPlan1d(&plan, size, HIPFFT_C2C, 1));
	handle_fft(hipfftExecC2C(plan, (hipfftComplex *) device_input, (hipfftComplex *) device_output, HIPFFT_FORWARD));
	handle_error(hipMemcpy(host_output, device_output, sizeof(float2) * size, hipMemcpyDeviceToHost));
	for (i = 0; i < size; i++)
	{
		output[i] = sqrt(host_output[i].x * host_output[i].x + host_output[i].y * host_output[i].y);
	}
	handle_error(hipHostFree(host_input));
	handle_error(hipHostFree(host_output));
	handle_error(hipFree(device_input));
	handle_error(hipFree(device_output));
	handle_fft(hipfftDestroy(plan));
	return 0;
}
